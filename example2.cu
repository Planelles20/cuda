#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void Multy2Matrix(int m, int n, int k, float* A, float* B, float* C)
{
  int Row = blockIdx.y*blockDim.y+threadIdx.y;
  int Col = blockIdx.x*blockDim.x+threadIdx.x;
  if ((Row < n) && (Col < k)) {
    float Cvalue = 0.0;
    for (int i = 0; i < m; ++i)  Cvalue += A[Row*m+i] * B[Col+i*k];
    C[Row*k+Col] = Cvalue;
  }
}
/*
  A(NxM) x B(MxK) = C(NxK)
*/

int main(void)
{
  float *d_a, *d_b, *d_c;
  float *h_a, *h_b, *h_c;

  const int N = 9;
  const int M = 5;
  const int K = 3;

  size_t size_a = N * M * sizeof(float);
  size_t size_b = M * K * sizeof(float);
  size_t size_c = N * K * sizeof(float);

  h_a = (float *)malloc(size_a);
  h_b = (float *)malloc(size_b);
  h_c = (float *)malloc(size_c);
  hipMalloc((void **) &d_a, size_a);
  hipMalloc((void **) &d_b, size_b);
  hipMalloc((void **) &d_c, size_c);

  for (int j=0; j<N; j++){
    for (int i=0; i<M; i++){
      h_a[j*M+i] = i+j;
    }
  }

  for (int j=0; j<M; j++){
    for (int i=0; i<K; i++){
      h_b[j*K+i] = (i<j ? j:i);
    }
  }

  hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);

  // el numero de threads por bloque debe ser menor que 1024
  // en mi caso
  // threads x threads >= 1024
  const int threads = 20; //20 x 20 = 400

  dim3 DimGrid((N-1)/threads + 1, (K-1)/threads + 1, 1);
  dim3 DimBlock(threads, threads, 1);
  Multy2Matrix <<< DimGrid, DimBlock >>> (M, N, K, d_a, d_b, d_c);

  hipMemcpy(h_c, d_c, sizeof(float)*N*K, hipMemcpyDeviceToHost);
  std::cout << "B(KxM) x A(MxN) = C(KxN)\n";
  std::cout << "\nA matrix \n";
  for (int i=0; i<M; i++){
    for (int j=0; j<N; j++){
      std::cout << h_a[j*M+i] << " ";
    }
    std::cout << "\n";
  }

  std::cout << "\nB matrix \n";
  for (int i=0; i<K; i++){
    for (int j=0; j<M; j++){
      std::cout << h_b[j*K+i] << " ";
    }
    std::cout << "\n";
  }

  std::cout << "\nC matrix \n";
  for (int i=0; i<K; i++){
    for (int j=0; j<N; j++){
      std::cout << h_c[j*K+i] << " ";
    }
    std::cout << "\n";
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

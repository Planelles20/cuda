#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void Multy2Matrix(int m, int n, int k, float* A, float* B, float* C)
{
  int Row = blockIdx.y*blockDim.y+threadIdx.y;
  int Col = blockIdx.x*blockDim.x+threadIdx.x;
  if ((Row < n) && (Col < k)) {
    float Cvalue = 0.0;
    for (int i = 0; i < n; ++i)  Cvalue += A[Row*n+i] * B[Col+i*m];
    C[Row*n+Col] = Cvalue;
  }
}

/*
  A(NxM) x B(MxK) = C(NxK)
*/

int main(void)
{
  float *d_a, *d_b, *d_c;
  float *h_a, *h_b, *h_c;

  const int N = 4;
  const int M = 5;
  const int K = 4;

  size_t size_a = N * M * sizeof(float);
  size_t size_b = M * K * sizeof(float);
  size_t size_c = N * K * sizeof(float);

  h_a = (float *)malloc(size_a);
  h_b = (float *)malloc(size_b);
  h_c = (float *)malloc(size_c);
  hipMalloc((void **) &d_a, size_a);
  hipMalloc((void **) &d_b, size_b);
  hipMalloc((void **) &d_c, size_c);

  for (int i=0; i<N; i++){
    for (int j=0; j<M; j++){
      h_a[j*M+i] = i+j;
    }
  }

  for (int i=0; i<M; i++){
    for (int j=0; j<K; j++){
      h_b[j*K+i] = (i<j ? j:i);
    }
  }

  hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);

  dim3 DimGrid((N-1)/16 + 1, (K-1)/16 + 1, 1);
  dim3 DimBlock(16, 16, 1);
  Multy2Matrix <<< DimGrid, DimBlock >>> (M, N, K, d_a, d_b, d_c);

  hipMemcpy(h_c, d_c, sizeof(float)*N*K, hipMemcpyDeviceToHost);

  std::cout << "\nA matrix \n";
  for (int i=0; i<N; i++){
    for (int j=0; j<M; j++){
      std::cout << h_a[j*M+i] << " ";
    }
    std::cout << "\n";
  }

  std::cout << "\nB matrix \n";
  for (int i=0; i<M; i++){
    for (int j=0; j<K; j++){
      std::cout << h_b[j*K+i] << " ";
    }
    std::cout << "\n";
  }

  std::cout << "\nC matrix \n";
  for (int i=0; i<N; i++){
    for (int j=0; j<K; j++){
      std::cout << h_c[j*K+i] << " ";
    }
    std::cout << "\n";
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);



}
